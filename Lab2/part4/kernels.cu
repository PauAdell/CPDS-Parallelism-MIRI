#include <math.h>
#include <float.h>
#include <hip/hip_runtime.h>

__global__ void gpu_Heat (float *h, float *g, int N) {

	int i = blockIdx.x * blockDim.x + threadIdx.x;
    	int j = blockIdx.y * blockDim.y + threadIdx.y;

    	if( i > 0 && i < (N-1) && j > 0 && j < (N-1) ) { // we don't want to compute the edges
    		g[N * i + j] = 0.25f *(
			h[N * (i + 1) + j] +
			h[N * (i - 1) + j] +
			h[N * i + j - 1] +
			h[N * i + j + 1]);
	}
}


__global__ void gpu_Diff(float *u, float *utmp, float* diffs, int N) {
    int i = (blockIdx.y * blockDim.y) + threadIdx.y;
    int j = (blockIdx.x * blockDim.x) + threadIdx.x;

    if (i > 0 && i < N-1 && j > 0 && j < N-1){
        utmp[i*N+j]= 0.25 * (u[ i*N     + (j-1) ]+  // left
                u[ i*N     + (j+1) ]+  // right
                u[ (i-1)*N + j     ]+  // top
                u[ (i+1)*N + j     ]); // bottom
        diffs[i*N+j] = utmp[i*N+j] - u[i*N+j];
        diffs[i*N+j] *= diffs[i*N+j];
    }
}

__global__ void gpu_Heat_reduction(float *idata, float *odata, int N) {
	extern __shared__ float sdata[];
	unsigned int s;

	unsigned int tid = threadIdx.x;
	unsigned int i = blockIdx.x * (blockDim.x * 2) + threadIdx.x;
	unsigned int gridSize = blockDim.x * 2 * gridDim.x;
	sdata[tid] = 0;
	while (i < N) {
		sdata[tid] += idata[i] + idata[i + blockDim.x];
		i += gridSize;
	}
	__syncthreads();

	for (s = blockDim.x / 2; s > 32; s >>= 1) {
		if (tid < s)
			sdata[tid] += sdata[tid + s];
		__syncthreads();
	}
	if (tid < 32) {
		volatile float *smem = sdata;

		smem[tid] += smem[tid + 32];
		smem[tid] += smem[tid + 16];
		smem[tid] += smem[tid + 8];
		smem[tid] += smem[tid + 4];
		smem[tid] += smem[tid + 2];
		smem[tid] += smem[tid + 1];
	}

	if (tid == 0)
		odata[blockIdx.x] = sdata[0];
}
